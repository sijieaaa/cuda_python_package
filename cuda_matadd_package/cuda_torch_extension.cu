#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void add_kernel(float* a, float* b, float* c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

void add_cuda(torch::Tensor a, torch::Tensor b, torch::Tensor c) {
    int N = a.numel();
    const int threads = 1024;
    const int blocks = (N + threads - 1) / threads;
    add_kernel<<<blocks, threads>>>(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(), N);
}

// PyBind11 
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("add", &add_cuda, "Matrix addition using CUDA");
}
